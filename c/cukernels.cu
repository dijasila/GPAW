#include "hip/hip_runtime.h"
#include <complex.h>
#include <stdio.h>
#include <hipblas.h> 
#include <hip/hip_math_constants.h>

__global__ void add( hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N)
    c[tid] = hipCadd(a[tid], b[tid]);
}

__global__ void mul( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(a[tid], b[tid]);
}

__global__ void mulc( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(hipConj(a[tid]), b[tid]);
}

__global__ void map_G2Q( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[c[tid]] = a[tid];
}

__global__ void map_Q2G( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[tid] = a[c[tid]];
}

__global__ void trans_wfs( hipDoubleComplex *a, hipDoubleComplex *b, int *index, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[index[tid]] = a[tid]; /*hipCmul(a[tid], phase[tid]);*/
}

__global__ void trans_wfs_noindex( hipDoubleComplex *a, hipDoubleComplex *b, int *C, double *dk, int ng0, int ng1, int ng2 ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int n = ng0 * ng1 * ng2;
  if (tid < n) {
    int g2 = tid % ng2;
    int g1 = (tid / ng2) % ng1;
    int g0 = (tid / ng2) / ng1;

    int p0 = ((C[0] * g0 + C[3] * g1 + C[6] * g2) % ng0 + ng0) % ng0;
    int p1 = ((C[1] * g0 + C[4] * g1 + C[7] * g2) % ng1 + ng1) % ng1;
    int p2 = ((C[2] * g0 + C[5] * g1 + C[8] * g2) % ng2 + ng2) % ng2;

    int index = (p0 * ng1 + p1) * ng2 + p2;
    double tmp = dk[0]/ng0*p0 + dk[1]/ng1*p1 + dk[2]/ng2*p2; 
    hipDoubleComplex phase  = make_hipDoubleComplex(cos(2*HIP_PI*tmp), sin(2*HIP_PI*tmp));

    b[index] = hipCmul(a[tid], phase); 
  }

}


__global__ void conj( hipDoubleComplex *a, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) a[tid] = hipConj(a[tid]);
}

__global__ void copy( hipDoubleComplex *a, hipDoubleComplex *b, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) b[tid] = a[tid];
}



__global__ void P_ai( double *spos_ac, double *ibzk_kc, int *op_scc, int *a_sa,
		      double **R_asii, hipDoubleComplex **P_ani, hipDoubleComplex **Pout_ai, int *Ni_a, bool time_rev, 
		      int Na, int s, int ik, int n){
  int tid = threadIdx.x;
  int ia = blockIdx.x;
  __shared__ hipDoubleComplex x;
  __shared__ double S_c[3];
  __shared__ double tmp;

  x=make_hipDoubleComplex(0., 0.);
  tmp = 0. ;

  int ib=a_sa[s*Na+ia];
  int Ni=Ni_a[ia];

  if (tid < 3){
    S_c[tid] = 0.;
    for (int dim=0; dim<3; dim++){
      S_c[tid] += spos_ac[ia*3+dim] * op_scc[s*9+dim*3+tid] ;
     }
    S_c[tid] -= spos_ac[ib*3+tid];
  }

  __syncthreads();

  tmp = S_c[0] * ibzk_kc[ik*3+0] + S_c[1] * ibzk_kc[ik*3+1] + S_c[2] * ibzk_kc[ik*3+2];

  x = make_hipDoubleComplex(cos(2*HIP_PI*tmp), sin(2*HIP_PI*tmp));

  if (tid < Ni){
    for (int j=0; j<Ni; j++){
      Pout_ai[ia][tid] = hipCadd(hipCmul(make_hipDoubleComplex(R_asii[ia][s*Ni*Ni+tid*Ni+j],0), P_ani[ib][n*Ni+j]), Pout_ai[ia][tid]);
      __syncthreads();
    }
    Pout_ai[ia][tid] = hipCmul(x, Pout_ai[ia][tid]);

    if (time_rev > 0){
      Pout_ai[ia][tid] = hipConj(Pout_ai[ia][tid]);
    }
  }
}



extern "C" {
void cudaAdd( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  add<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMul( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mul<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMulc( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mulc<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMap_G2Q( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  map_G2Q<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N);
}
}

extern "C" {
void cudaMap_Q2G( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  map_Q2G<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N);
}
}

extern "C" {
  void cudaTransform_wfs( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
    int threads = 128;
    int blocks = N/threads + (N%threads == 0 ? 0:1);
    trans_wfs<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N );
  }
}

extern "C" {
  void cudaTransform_wfs_noindex( double complex* dev_a, double complex* dev_b, int* dev_c, double* dk, int N0, int N1, int N2 ) {
    int threads = 128;
    int N = N0 * N1 * N2;
    int blocks = N/threads + (N%threads == 0 ? 0:1);
    trans_wfs_noindex<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, (double*)dk, N0, N1, N2 );
  }
}

extern "C" {
  void cudaConj( double complex* dev_a, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  conj<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, N);
}
}

extern "C" {
  void cudaCopy( double complex* dev_a, double complex* dev_b, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  copy<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, N);
}
}


extern "C" {
  void cudaP_ai( double* dev_spos_ac, double* dev_ibzk_kc, int* dev_op_scc, int* dev_a_sa, 
		 double **dev_R_asii, double complex **P_ani, double complex **Pout_ai, int* Ni_a,
		 bool time_rev, int Na, int s, int ik, int n){

  int threads = 128;
  int blocks = Na;

  P_ai<<<blocks, threads>>>( (double*)dev_spos_ac, (double*)dev_ibzk_kc, (int*)dev_op_scc, (int*)dev_a_sa, 
			     (double**)dev_R_asii,(hipDoubleComplex**)P_ani, (hipDoubleComplex**)Pout_ai, (int*)Ni_a,
			     time_rev, Na, s, ik, n);
}
}


