#include "hip/hip_runtime.h"
#include <complex.h>
#include <stdio.h>
#include <hipblas.h> 
#include <hip/hip_math_constants.h>

__global__ void add( hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N)
    c[tid] = hipCadd(a[tid], b[tid]);
}

__global__ void mul( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(a[tid], b[tid]);
}

__global__ void mulc( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(hipConj(a[tid]), b[tid]);
}

__global__ void map_G2Q( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n, int nG0, int nmultix ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n*nmultix) b[c[tid%n]+tid/n*nG0] = a[tid];
}

__global__ void map_Q2G( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n, int nG0, int nmultix ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n*nmultix) b[tid] = a[c[tid%n]+tid/n*nG0];
}

__global__ void density_matrix_R( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int n, int nmultix ){
  /* perform psit1_R.conj() * expqr_R * psit2_uR */
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int ii = tid%n;
  if (tid < n*nmultix) {
    c[tid] = hipCmul(hipCmul(hipConj(a[ii]), b[ii]), c[tid]);
  }
}



__global__ void trans_wfs( hipDoubleComplex *a, hipDoubleComplex *b, int *index, int n, int nmultix ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n*nmultix) b[index[tid%n]+tid/n*n] = a[tid]; /*hipCmul(a[tid], phase[tid]);*/
}

__global__ void trans_wfs_noindex( hipDoubleComplex *a, hipDoubleComplex *b, int *C, double *dk, int ng0, int ng1, int ng2 ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int n = ng0 * ng1 * ng2;
  if (tid < n) {
    int g2 = tid % ng2;
    int g1 = (tid / ng2) % ng1;
    int g0 = (tid / ng2) / ng1;

    int p0 = ((C[0] * g0 + C[3] * g1 + C[6] * g2) % ng0 + ng0) % ng0;
    int p1 = ((C[1] * g0 + C[4] * g1 + C[7] * g2) % ng1 + ng1) % ng1;
    int p2 = ((C[2] * g0 + C[5] * g1 + C[8] * g2) % ng2 + ng2) % ng2;

    int index = (p0 * ng1 + p1) * ng2 + p2;
    double tmp = dk[0]/ng0*p0 + dk[1]/ng1*p1 + dk[2]/ng2*p2; 
    hipDoubleComplex phase  = make_hipDoubleComplex(cos(2*HIP_PI*tmp), sin(2*HIP_PI*tmp));

    b[index] = hipCmul(a[tid], phase); 
  }

}


__global__ void conj( hipDoubleComplex *a, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) a[tid] = hipConj(a[tid]);
}

__global__ void copy( hipDoubleComplex *a, hipDoubleComplex *b, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) b[tid] = a[tid];
}



__global__ void P_ai( double *spos_ac, double *ibzk_kc, int *op_scc, int *a_sa,
		      double **R_asii, hipDoubleComplex **P_ani, hipDoubleComplex **Pout_ai, 
		      int *Ni_a, bool time_rev, 
		      int Na, int s, int ik, int n){
  int tid = threadIdx.x;
  int ia = blockIdx.x;
  __shared__ hipDoubleComplex x;
  __shared__ double S_c[3];
  __shared__ double tmp;

  x=make_hipDoubleComplex(0., 0.);
  tmp = 0. ;

  int ib=a_sa[s*Na+ia];
  int Ni=Ni_a[ia];

  if (tid < 3){
    S_c[tid] = 0.;
    for (int dim=0; dim<3; dim++){
      S_c[tid] += spos_ac[ia*3+dim] * op_scc[s*9+dim*3+tid] ;
     }
    S_c[tid] -= spos_ac[ib*3+tid];
  }

  __syncthreads();

  tmp = S_c[0] * ibzk_kc[ik*3+0] + S_c[1] * ibzk_kc[ik*3+1] + S_c[2] * ibzk_kc[ik*3+2];

  x = make_hipDoubleComplex(cos(2*HIP_PI*tmp), sin(2*HIP_PI*tmp));

  if (tid < Ni){
    for (int j=0; j<Ni; j++){
      Pout_ai[ia][tid] = hipCadd(hipCmul(make_hipDoubleComplex(R_asii[ia][s*Ni*Ni+tid*Ni+j],0), P_ani[ib][n*Ni+j]), Pout_ai[ia][tid]);
      __syncthreads();
    }
    Pout_ai[ia][tid] = hipCmul(x, Pout_ai[ia][tid]);

    if (time_rev > 0){
      Pout_ai[ia][tid] = hipConj(Pout_ai[ia][tid]);
    }
  }
}


__global__ void Q_anL(hipDoubleComplex **P1_ami, hipDoubleComplex **P2_ai, 
		      double **Delta_apL, hipDoubleComplex **Q_amL, int mband, int Na, 
		      int *Ni_a, int *nL_a){
  int tidx = threadIdx.x;
  int tidy = threadIdx.y;  
  int ia = blockIdx.x;

  int Ni=Ni_a[ia];
  int nL=nL_a[ia];

  if (blockDim.x < mband) printf("Q_anL calculation is wrong !! ");
  if (blockDim.y < nL) printf("Q_anL calculation is wrong !! ");

  if (tidx < mband && tidy < nL){
    for (int ix=0; ix<Ni; ix++){
      for (int iy=0; iy<Ni; iy++){
        int ip = ix*Ni+iy;
	hipDoubleComplex tmp = hipCmul(P1_ami[ia][tidx*Ni+ix], P2_ai[ia][iy]);
	hipDoubleComplex tmp2 = make_hipDoubleComplex(Delta_apL[ia][ip*nL+tidy] * hipCreal(tmp), 
			     Delta_apL[ia][ip*nL+tidy] * hipCimag(tmp));
        Q_amL[ia][tidx*nL+tidy] = hipCadd(Q_amL[ia][tidx*nL+tidy], tmp2);
      }
    }
  }

  /*  __shared__ hipDoubleComplex Q_mL;
      Q_mL = Q_amL[ia];*/

  /*  
  if (tidx < Ni && tidy < Ni){
    for (int im=0; im<mband; im++){
      hipDoubleComplex tmp = hipCmul(P1_ami[ia][im*Ni+tidx], P2_ai[ia][tidy]);
      for (int i=0; i<nL; i++){
        int ip=tidx+Ni+tidy;
	hipDoubleComplex tmp2 = make_hipDoubleComplex(Delta_apL[ia][ip*nL+i] * hipCreal(tmp), 
			     Delta_apL[ia][ip*nL+i] * hipCimag(tmp));
	atomicAdd( &(Q_amL[ia][im*nL+i]), tmp2);
      }
    }
  }

  __syncthreads();
*/
}


extern "C" {
void cudaAdd( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  add<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMul( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mul<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMulc( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mulc<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
  void cudaMap_G2Q( double complex* dev_a, double complex* dev_b, int* dev_c, int N, int nG0, int nmultix ) {
  int threads = 128;
  int nn = N * nmultix;
  int blocks = nn/threads + (nn%threads == 0 ? 0:1);
  map_G2Q<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N, nG0, nmultix);
}
}

extern "C" {
  void cudaMap_Q2G( double complex* dev_a, double complex* dev_b, int* dev_c, int N, int nG0, int nmultix ) {
  int threads = 128;
  int nn = N * nmultix;
  int blocks = nn/threads + (nn%threads == 0 ? 0:1);
  map_Q2G<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N, nG0, nmultix);
}
}

extern "C" {
  void cudaDensity_matrix_R( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N, int nmultix ) {
  int threads = 128;
  int nn = N * nmultix;
  int blocks = nn/threads + (nn%threads == 0 ? 0:1);
  density_matrix_R<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N, nmultix);
}
}


extern "C" {
  void cudaTransform_wfs( double complex* dev_a, double complex* dev_b, int* dev_c, int N, int nmultix ) {
    int threads = 128;
    int nn = N * nmultix;
    int blocks = nn/threads + (nn%threads == 0 ? 0:1);
    trans_wfs<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N, nmultix );
  }
}

extern "C" {
  void cudaTransform_wfs_noindex( double complex* dev_a, double complex* dev_b, int* dev_c, double* dk, int N0, int N1, int N2 ) {
    int threads = 128;
    int N = N0 * N1 * N2;
    int blocks = N/threads + (N%threads == 0 ? 0:1);
    trans_wfs_noindex<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, (double*)dk, N0, N1, N2 );
  }
}

extern "C" {
  void cudaConj( double complex* dev_a, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  conj<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, N);
}
}

extern "C" {
  void cudaCopy( double complex* dev_a, double complex* dev_b, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  copy<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, N);
}
}


extern "C" {
  void cudaP_ai( double* dev_spos_ac, double* dev_ibzk_kc, int* dev_op_scc, int* dev_a_sa, 
		 double **dev_R_asii, double complex **P_ani, double complex **Pout_ai, int* Ni_a,
		 bool time_rev, int Na, int s, int ik, int n){

  int threads = 128;
  int blocks = Na;

  P_ai<<<blocks, threads>>>( (double*)dev_spos_ac, (double*)dev_ibzk_kc, 
			     (int*)dev_op_scc, (int*)dev_a_sa, 
			     (double**)dev_R_asii,(hipDoubleComplex**)P_ani, 
			     (hipDoubleComplex**)Pout_ai, (int*)Ni_a,
			     time_rev, Na, s, ik, n);
}
}



extern "C" {
  void cudaQ_anL( double complex **P1_ami, double complex **P2_ai,  
		  double **Delta_apL, double complex **Q_amL, int mband, int Na, 
		  int* Ni_a, int* nL_a){

  dim3 threads(16,16);
  int blocks = Na;

  Q_anL<<<blocks, threads>>>( (hipDoubleComplex**)P1_ami, (hipDoubleComplex**)P2_ai,
			     (double**)Delta_apL, (hipDoubleComplex**)Q_amL,
			     mband, Na, (int*)Ni_a, (int*)nL_a);

}
}

