#include "hip/hip_runtime.h"
#include "hipblas.h" 
#include "complex.h"
#include <stdio.h>
#include <"hip/hip_math_constants.h">

__global__ void add( hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N)
    c[tid] = hipCadd(a[tid], b[tid]);
}

__global__ void mul( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(a[tid], b[tid]);
}

__global__ void mulc( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(hipConj(a[tid]), b[tid]);
}

__global__ void map_G2Q( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[c[tid]] = a[tid];
}

__global__ void map_Q2G( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[tid] = a[c[tid]];
}

__global__ void trans_wfs( hipDoubleComplex *a, hipDoubleComplex *b, int *index, hipDoubleComplex *phase, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[index[tid]] = hipCmul(a[tid], phase[tid]);
}

__global__ void conj( hipDoubleComplex *a, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) a[tid] = hipConj(a[tid]);
}

__global__ void copy( hipDoubleComplex *a, hipDoubleComplex *b, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) b[tid] = a[tid];
}

__global__ void P_ai( double *spos1_c, double *spos2_c, double *k_c, int *op_cc, 
		      hipDoubleComplex *R_ii, hipDoubleComplex *Pin_i, hipDoubleComplex *Pout_i, int Ni){
  int tid = threadIdx.x;
  double complex x=0;
  double S_c[3] = [0,0,0];

  if (tid < 3){
    for (int dim=0; dim<3; dim++){
      S_c[tid] += spos1_c[dim] * op_cc[dim*3+tid] ;
      __syncthreads();
    }
    S_c[tid] -= spos2_c[tid];

    x += cos(2*HIP_PI*S_c[tid] * k_c[tid]) + I * sin(2*HIP_PI*S_c[tid] * k_c[tid]);
  }
  __syncthreads();

  if (tid < Ni){
    for (int j=0; j<3; j++){
      Pout_i[tid] += R_ii[tid*Nj+j] * Rin_i[j];
      _syncthreads();
    }
    Pout_i[tid] *= x;
  }


}


extern "C" {
void cudaAdd( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  add<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMul( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mul<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMulc( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mulc<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMap_G2Q( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  map_G2Q<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N);
}
}

extern "C" {
void cudaMap_Q2G( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  map_Q2G<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N);
}
}

extern "C" {
  void cudaTransform_wfs( double complex* dev_a, double complex* dev_b, int* dev_c, double complex* dev_d, int N ) {
    int threads = 128;
    int blocks = N/threads + (N%threads == 0 ? 0:1);
    trans_wfs<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, (hipDoubleComplex*)dev_d, N );
  }
}

extern "C" {
  void cudaConj( double complex* dev_a, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  conj<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, N);
}
}

extern "C" {
  void cudaCopy( double complex* dev_a, double complex* dev_b, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  copy<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, N);
}
}

extern "C" {
  void cudaP_ai( double* dev_spos_c, int* dev_op_cc, double* dev_S_c) {
  int threads = 128;
  int blocks = 1;
  P_ai<<<blocks, threads>>>( dev_spos_c, dev_op_cc, dev_S_c);
}
}



