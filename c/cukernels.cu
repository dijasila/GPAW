#include "hip/hip_runtime.h"
#include "hipblas.h" 
#include "complex.h"
#include <stdio.h>

__global__ void add( hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N)
    c[tid] = hipCadd(a[tid], b[tid]);
}

__global__ void mul( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(a[tid], b[tid]);
}

__global__ void mulc( hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c, int N ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) c[tid] = hipCmul(hipConj(a[tid]), b[tid]);
}

__global__ void map_G2Q( hipDoubleComplex *a, hipDoubleComplex *b, int *c, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[c[tid]] = a[tid];
}

__global__ void trans_wfs( hipDoubleComplex *a, hipDoubleComplex *b, int *index, hipDoubleComplex *phase, int n ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) b[index[tid]] = hipCmul(a[tid], phase[tid]);
}




extern "C" {
void cudaAdd( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  add<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMul( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mul<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMulc( double complex* dev_a, double complex* dev_b, double complex* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  mulc<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (hipDoubleComplex*)dev_c, N);
}
}

extern "C" {
void cudaMap_G2Q( double complex* dev_a, double complex* dev_b, int* dev_c, int N ) {
  int threads = 128;
  int blocks = N/threads + (N%threads == 0 ? 0:1);
  map_G2Q<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, N);
}
}

extern "C" {
  void cudaTransform_wfs( double complex* dev_a, double complex* dev_b, int* dev_c, double complex* dev_d, int N ) {
    int threads = 128;
    int blocks = N/threads + (N%threads == 0 ? 0:1);
    trans_wfs<<<blocks, threads>>>( (hipDoubleComplex*)dev_a, (hipDoubleComplex*)dev_b, (int*)dev_c, (hipDoubleComplex*)dev_d, N );
  }
}
